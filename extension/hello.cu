
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__global__ void add(int *a,int *b,int *c){
	int t=blockIdx.x;
	if(t<N)
		c[t]=gridDim.x;
}

int main(){
	int a[N],b[N],c[N];
	int *a_cuda,*b_cuda,*c_cuda;
	//赋值
	for(int i=0;i<N;i++){
		a[i]=i-3;
		b[i]=i/2+1;
	}
	hipMalloc((void**)&a_cuda,N*sizeof(int));
	hipMalloc((void**)&b_cuda,N*sizeof(int));
	hipMalloc((void**)&c_cuda,N*sizeof(int));
	hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b_cuda,b,N*sizeof(int),hipMemcpyHostToDevice);
	add<<<N, 1>>>(a_cuda,b_cuda,c_cuda);
	hipMemcpy(c,c_cuda,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("a+b=(");
	for(int i=0;i<N;i++)
		printf("%d,",c[i]);
	printf(")\n");
	hipFree(a_cuda);
	hipFree(b_cuda);
	hipFree(c_cuda);
}